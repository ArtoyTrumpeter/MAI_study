
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#define CSC(call) 						\
do {									\
	hipError_t	status = call;			\
	if (status != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));			\
		exit(0);						\
	}									\
} while(0)

__global__ void kernel(int *arr, int n) {
	// threadIdx.x, threadIdx.y, threadIdx.z,
	// blockIdx.x, blockIdx.y, blockIdx.z,
	// blockDim.x, blockDim.y, blockDim.z,
	// gridDim.x, gridDim.y, gridDim.z,
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	while (idx < n) {
	//	assert(idx < n - 10);
		arr[idx] *= 2;
		idx += offset;
	}
}

int main() {
	int i, n = 10000000;
	scanf("%d", &n);
	int *arr = (int *)malloc(sizeof(int) * n);
	for(i = 0; i < n; i++)
		arr[i] = i;
	int *dev_arr;
	CSC(hipMalloc(&dev_arr, sizeof(int) * n));

	CSC(hipMemcpy(dev_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice));

	hipEvent_t start, end;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));

	kernel<<<256, 256>>>(dev_arr, n);
	CSC(hipGetLastError());

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));

	printf("time = %f\n", t);

	CSC(hipMemcpy(arr, dev_arr, sizeof(int) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(dev_arr));
	for(i = n - 10; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");
	return 0;
}