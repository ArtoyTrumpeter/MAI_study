#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/scan.h>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)                                                          \

__global__ void histogram(int* dev_hist, int* dev_arr, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
    while (idx < n) {
		atomicAdd(dev_hist + dev_arr[idx], 1);
		idx += offset;
	}
}

__global__ void out(int* dev_hist, int* dev_arr, int* dev_out, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = idx; i < n; i += offset) {
		dev_out[atomicAdd(dev_hist + dev_arr[i], -1) - 1] = dev_arr[i];
	}
}

void counting_sort(int* dev_arr, int* dev_hist, int* dev_out, int n, int max) {
	histogram<<<1024, 1024>>>(dev_hist, dev_arr, n);
	CSC(hipGetLastError());
	
	thrust::device_ptr<int> ptr = thrust::device_pointer_cast(dev_hist);
    thrust::inclusive_scan(ptr, ptr + max + 1, ptr);
	
	out<<<1024, 1024>>>(dev_hist, dev_arr, dev_out, n);
	CSC(hipGetLastError());
}

int maximum(int* arr, int n) {
    int max = 0;
    for (int i = 0; i < n; i++) {
        if (arr[i] > max) {
            max = arr[i];
        }
    }
    return max;
}

int main() {
	int n;
	fread(&n, sizeof(int), 1, stdin);
	int* arr = (int*) malloc(sizeof(int) * n);
    fread(arr, sizeof(int), n, stdin);
	int max = maximum(arr, n);

    int* dev_arr;
	CSC(hipMalloc(&dev_arr, sizeof(int) * n));
	CSC(hipMemcpy(dev_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice));

	int* dev_out;
	CSC(hipMalloc(&dev_out, sizeof(int) * n));

	int* dev_hist;
	CSC(hipMalloc(&dev_hist, sizeof(int) * (max + 1)));
	CSC(hipMemset(dev_hist, 0, sizeof(int) * (max + 1)));

	hipEvent_t start, end;

	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end));
	CSC(hipEventRecord(start));
	
	counting_sort(dev_arr, dev_hist, dev_out, n, max);

	CSC(hipEventRecord(end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));
	printf("kernel = <<<64, 64>>>, time = %f\n", t);

	CSC(hipMemcpy(arr, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost));

	//fwrite(arr, sizeof(int), n, stdout);

    CSC(hipFree(dev_arr));
    CSC(hipFree(dev_hist));
    CSC(hipFree(dev_out));
	free(arr);
    
    return 0;
}